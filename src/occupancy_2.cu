/* 
 *  
 *  New York University
 *  GPUs Fall 2017
 *  Steven Adam & Michael Corso
 *  Quantifying the Relationship Between Occupancy and Performance
 * 
 *
 *  Code Explanation:
 *
 *      (1) The first function call is to initDeviceVars() which checks the system for CUDA devices and
 *      chooses the device with the highest Compute Capability (CC). Device-specific variables are then 
 *      populated based on the CC and variables stored in the cudaDeviceProp struct. This information
 *      allows for grid and block dimensions to be constructed according to the specific device this code
 *      is running on.
 *
 *      (2) The code then scales the kernel's parameters based on the device's specifications using the 
 *		user-specified values provided in the program's arguments:
 *
 *          (a) occupancyMethod: 
 *
 *              (0) Blocks per SM: Determines the maximum number of blocks assignable (IE [number of SMs] *
 *              [max blocks assignable to each SM]) and scales it based on the specified targetOccupancy.
 *              The number of threads per block is maxed.
 *
 *              (1) Threads per Block: Determines the maximum number of threads assignable to a block (IE
 *              1024 is common) and scales this based on the specified targetOccupancy. The number of blocks
 *              is equal to ([number of SMs] * [max blocks assignable to each SM]).
 *
 *              (2) Inverted Blocks per Grid to Threads per Block: This combines the previous two tests by
 *              scaling the number of blocks simultaneously assignable (see (1) above) and the max number of
 *              threads per block (see (2) above). These values are inversely scaled based on the specified
 *              targetOccupancy (IE Specifying a 75% occupancy will set Blocks per SM to 75% of capacity and
 *              Threads per Block will be set to 25% of capacity).
 *
 *      	(b) The work being performed by the threads is user-specified in the program's arguments:
 *      
 *          	(0) doubleInt(): 	No memory accesses, simply multiples its thread id by 2
 *
 *          	(1) vectorMaths1(): Memory-bound vector addition, 3 memory accesses, 1 floating-point addition
 *									CGMA = 1/3
 *              
 *          	(2) vectorMaths2(): Compute-bound vector math, 3 memory accesses, 90 floating-point operations
 *									CGMA = 90/3 = 30
 *
 *          (c) targetOccupancy: An integer value of 1 - 100 which specifies the percentage of the maximum 
 *              occupancy for this test.
 *              
 *          (d) problemSize: An integer value which specifies the amount of work to be performed by the kernel
 *              
 *
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
#include <algorithm>
#include <vector>

// DEBUG/TEST
#define TESTING false
#define PRINTTIME true

//
void howToUse();

// OCCUPANCY FUNCTIONS
double test_BlocksPerSM();
double test_ThreadsPerBlock();
double test_ThreadsPerBlockPerKernel();

// GPU SPEC FUNCTIONS
void initDeviceVars();
void getGPU();
void getMaxBlocksPerSM();
void getMaxWarpsPerSM();

// GPU FUNCTIONS
__global__
void doubleInt (int, int);
__global__
void vectorMaths1 (float *, float *, float *, int, int);
__global__
void vectorMaths2 (float *, float *, float *, int, int);

// TEST VARIABLES
int problemSize, occupancyMethod, functionToUse;
double targetOccupancy;

// DEVICE VARIABLES
char * deviceName;
int maxThreadsPerBlock, 
	maxThreadsPerSM, maxBlocksPerSM, maxWarpsPerSM, numSMs, 
	maxThreadsPerGrid, 
	compCapMajor, compCapMinor;

int main(int argc, char * argv[]) {

	// HOW TO USE
    if(argc != 5) howToUse();

	// UPDATE DEVICE VARIABLES
	initDeviceVars();

    // OUTPUT DEVICE-SPECIFIC VALUES
    if (TESTING) {
        printf("\nGPU Info:\n\t%-15s %s\n\t%-15s %d.%d\n\t%-15s %d\n\t%-15s %d\n\t%-15s %d\n\t%-15s %d\n", 
            "Device ID", deviceName,
            "Compute C.", compCapMajor, compCapMinor, 
            "Grid Size", maxThreadsPerGrid,
            "Block Size", maxThreadsPerBlock,
            "# SMs", numSMs,
            "# Warps", maxWarpsPerSM
            );
    }

    // GET USER-SPECIFIED VARIABLES 
    occupancyMethod = (int) atoi(argv[1]);
    functionToUse = (int) atoi(argv[2]);
    targetOccupancy = ((double) (atoi(argv[3]) / 100.0));
    if (targetOccupancy > 1.0) targetOccupancy = 1.0;
    if (targetOccupancy == 0.0) targetOccupancy = 0.01;
    problemSize = (int) atoi(argv[4]);
    
    // FOR STORING TIMES
    // std::vector<double> times(9);
    // times.clear();
    // for (int i2 = 0; i2 < 9; i2++) {

    // MAX BLOCKS THAT CAN RUN SIMULTANEOUSLY
    if (occupancyMethod == 0) {
        // times.push_back(test_BlocksPerSM());
        test_BlocksPerSM();
    }
    // MAX THREADS PER BLOCK
    else if (occupancyMethod == 1) {
        // times.push_back(test_ThreadsPerBlock());
        test_ThreadsPerBlock();
    }
    // THREADS/BLOCK INVERSED WITH BLOCKS/KERNEL
    else if (occupancyMethod == 2) {
        // times.push_back(test_ThreadsPerBlockPerKernel());
        test_ThreadsPerBlockPerKernel();
    }
    else {
        printf("\nNot an acceptable occupancyMethod!\n");
        howToUse();
    }

    // }
    // std::sort (times.begin(), times.end());
    // if (PRINTTIME) 
    //     printf("%d,%d,%.02f,%f\n", occupancyMethod, functionToUse, targetOccupancy, times.at(4));
    		
    return 0;
}

// BLOCKS PER SM / TOTAL BLOCKS IN THE KERNEL (USES MAX NUMBER OF THREADS PER BLOCK)
double test_BlocksPerSM() {

	// NUMBER OF BLOCKS
    int totalBlocks = ((numSMs * maxBlocksPerSM) * targetOccupancy);
    if (totalBlocks < 1) totalBlocks = 1;
    
    // ATTEMPT TO DISTRIBUTE THREADS EVENLY 
    int threadsPerBlock = (maxThreadsPerSM / (totalBlocks / numSMs));
    while (threadsPerBlock % 32 != 0) threadsPerBlock -= 1;
    if (threadsPerBlock < 128) threadsPerBlock = 128;
    if (threadsPerBlock > maxThreadsPerBlock) threadsPerBlock = maxThreadsPerBlock;

    // TOTAL NUMBER OF THREADS IN THE GRID
    int totalThreads = totalBlocks * threadsPerBlock;

    dim3 dimGrid(totalBlocks, 1, 1);                       
    dim3 dimBlock(threadsPerBlock, 1, 1);

    if (TESTING) printf("\ntest_MaxBlocksPerSM running with:\n\ttotalBlocks\t%d\t%d%%\n\tblockSize\t%d\t%.01f%%\n", 
        totalBlocks, ((int) (targetOccupancy * 100)), threadsPerBlock, (((float) threadsPerBlock / (float) maxThreadsPerBlock) * 100));

    // ARRAYS FOR PERFORMING VECTOR MATH
    float * in1 = (float *) calloc((problemSize), sizeof(float));
    float * in2 = (float *) calloc((problemSize), sizeof(float));
    float * out = (float *) calloc((problemSize), sizeof(float));
    float * in1D; float * in2D; float * outD;

    for (int i = 0; i < problemSize; i++) {
        in1[i] = (i * 0.99);
        in2[i] = ((problemSize - i - 1) * 0.99);
        out[i] = -1;
    }

    hipMalloc((void **) &in1D, problemSize);  
    hipMemcpy(in1D, in1, problemSize, hipMemcpyHostToDevice);   
    hipMalloc((void **) &in2D, problemSize);  
    hipMemcpy(in2D, in2, problemSize, hipMemcpyHostToDevice);  
    hipMalloc((void **) &outD, problemSize);  
    hipMemcpy(outD, out, problemSize, hipMemcpyHostToDevice);  

    // INITIALIZE TIMER BEFORE CALLING KERNEL
    clock_t start = clock();
    if (functionToUse == 0) {
        doubleInt<<<dimGrid, dimBlock>>>(problemSize, totalThreads);
    }
    else if (functionToUse == 1) {
        vectorMaths1<<<dimGrid, dimBlock>>>(in1D, in2D, outD, problemSize, totalThreads);
    }
    else if (functionToUse == 2) {
        vectorMaths2<<<dimGrid, dimBlock>>>(in1D, in2D, outD, problemSize, totalThreads);
    }

    // SYNC DEVICE AND GET TIME TAKEN
    hipDeviceSynchronize();
    clock_t end = clock();
    double time_taken = ((double)(end - start)) / CLOCKS_PER_SEC;

    // CLEANUP
    free(in1); free(in2); free(out);
    hipFree(in1D); hipFree(in2D); hipFree(outD);

    return time_taken;
}

// THREADS PER BLOCK (USES MAX NUMBER OF BLOCKS)
double test_ThreadsPerBlock() {

	// NUMBER OF THREADS PER BLOCK
    int threadsPerBlock = (maxThreadsPerBlock * targetOccupancy);
    while (threadsPerBlock % 32 != 0) threadsPerBlock -= 1;
    if (threadsPerBlock < 1) threadsPerBlock = 1;

    // NUMBER OF BLOCKS
    int totalBlocks = (maxThreadsPerSM / threadsPerBlock) * numSMs;
    if (totalBlocks < 1) totalBlocks = 1;

    // TOTAL NUMBER OF THREADS IN THE GRID
    int totalThreads = totalBlocks * threadsPerBlock;

    dim3 dimGrid(totalBlocks, 1, 1);                       
    dim3 dimBlock(threadsPerBlock, 1, 1);

    if (TESTING) printf("\ntest_ThreadsPerBlock running with:\n\ttotalBlocks\t%d\t100%%\n\tblockSize\t%d\t%d%%\n", 
        totalBlocks, threadsPerBlock, ((int) (targetOccupancy * 100)));

    // ARRAYS FOR PERFORMING VECTOR MATH
    float * in1 = (float *) calloc((problemSize), sizeof(float));
    float * in2 = (float *) calloc((problemSize), sizeof(float));
    float * out = (float *) calloc((problemSize), sizeof(float));
    float * in1D; float * in2D; float * outD;

    for (int i = 0; i < problemSize; i++) {
        in1[i] = (i * 0.99);
        in2[i] = ((problemSize - i - 1) * 0.99);
        out[i] = -1;
    }

    hipMalloc((void **) &in1D, problemSize);  
    hipMemcpy(in1D, in1, problemSize, hipMemcpyHostToDevice);   
    hipMalloc((void **) &in2D, problemSize);  
    hipMemcpy(in2D, in2, problemSize, hipMemcpyHostToDevice);  
    hipMalloc((void **) &outD, problemSize);  
    hipMemcpy(outD, out, problemSize, hipMemcpyHostToDevice);  

    // INITIALIZE TIMER BEFORE CALLING KERNEL
    clock_t start = clock();
    if (functionToUse == 0) {
        doubleInt<<<dimGrid, dimBlock>>>(problemSize, totalThreads);
    }
    else if (functionToUse == 1) {
        vectorMaths1<<<dimGrid, dimBlock>>>(in1D, in2D, outD, problemSize, totalThreads);
    }
    else if (functionToUse == 2) {
        vectorMaths2<<<dimGrid, dimBlock>>>(in1D, in2D, outD, problemSize, totalThreads);
    }

    // SYNC DEVICE AND GET TIME TAKEN
    hipDeviceSynchronize();
    clock_t end = clock();
    double time_taken = ((double)(end - start)) / CLOCKS_PER_SEC;

    // CLEANUP
    free(in1); free(in2); free(out);
    hipFree(in1D); hipFree(in2D); hipFree(outD);

    return time_taken;
}

// THREADS/BLOCK INVERSED WITH BLOCKS/KERNEL
// THIS ACTS LIKE A SEESAW: TOTALBLOCKS GOES UP AS THREADS PER BLOCK GOES DOWN AND VICE VERSA
double test_ThreadsPerBlockPerKernel() {

	// NUMBER OF BLOCKS
    int totalBlocks = ((numSMs * maxBlocksPerSM) * targetOccupancy);

    // THREADS PER BLOCK
    int threadsPerBlock = maxThreadsPerBlock * (1.0 - targetOccupancy);
    if (threadsPerBlock <= 0) threadsPerBlock = 1;
    if (totalBlocks <= 0) totalBlocks = 1;

    // TOTAL NUMBER OF THREADS IN THE GRID
    int totalThreads = totalBlocks * threadsPerBlock;

    dim3 dimGrid(totalBlocks, 1, 1);                       
    dim3 dimBlock(threadsPerBlock, 1, 1);

    if (TESTING) printf("\ntest_ThreadsPerBlockPerKernel running with:\n\ttotalBlocks\t%d\t%d%%\n\tblockSize\t%d\t%d%%\n", 
        totalBlocks, ((int) ceil((totalBlocks * 100.0) / (numSMs * maxBlocksPerSM))), 
        threadsPerBlock, ((int) ceil((threadsPerBlock * 100.0) / maxThreadsPerBlock)));

    // ARRAYS FOR PERFORMING VECTOR MATH
    float * in1 = (float *) calloc((problemSize), sizeof(float));
    float * in2 = (float *) calloc((problemSize), sizeof(float));
    float * out = (float *) calloc((problemSize), sizeof(float));
    float * in1D; float * in2D; float * outD;

    for (int i = 0; i < problemSize; i++) {
        in1[i] = (i * 0.99);
        in2[i] = ((problemSize - i - 1) * 0.99);
        out[i] = -1;
    }

    hipMalloc((void **) &in1D, problemSize);  
    hipMemcpy(in1D, in1, problemSize, hipMemcpyHostToDevice);   
    hipMalloc((void **) &in2D, problemSize);  
    hipMemcpy(in2D, in2, problemSize, hipMemcpyHostToDevice);  
    hipMalloc((void **) &outD, problemSize);  
    hipMemcpy(outD, out, problemSize, hipMemcpyHostToDevice);  

    // INITIALIZE TIMER BEFORE CALLING KERNEL
    clock_t start = clock();
    if (functionToUse == 0) {
        doubleInt<<<dimGrid, dimBlock>>>(problemSize, totalThreads);
    }
    else if (functionToUse == 1) {
        vectorMaths1<<<dimGrid, dimBlock>>>(in1D, in2D, outD, problemSize, totalThreads);
    }
    else if (functionToUse == 2) {
        vectorMaths2<<<dimGrid, dimBlock>>>(in1D, in2D, outD, problemSize, totalThreads);
    }

    // SYNC DEVICE AND GET TIME TAKEN
    hipDeviceSynchronize();
    clock_t end = clock();
    double time_taken = ((double)(end - start)) / CLOCKS_PER_SEC;

    // CLEANUP
    free(in1); free(in2); free(out);
    hipFree(in1D); hipFree(in2D); hipFree(outD);

    return time_taken;
}

// SIMPLE FUNCTION TO MAKE THE THREAD PERFORM WORK
// NO MEMORY ACCESS
// IF PROBLEM SIZE > NUMBER OF THREADS, THREADS WILL PERFORM MORE THAN ONE ACTION
__global__
void doubleInt (int N, int totalThreads) {

	int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    int val = id;

	while (id < N) {
        val = id;
        val *= 2;
        id += totalThreads;
	}
}

__global__
void vectorMaths1 (float * in1, float * in2, float * out, int N, int totalThreads) {

	int id = (blockIdx.x * blockDim.x) + threadIdx.x;

	while (id < N) {
		float t1 = in1[id];
		float t2 = in2[id];
		
		out[id] = t1 + t2;

        id += totalThreads;
	}
}

__global__
void vectorMaths2 (float * in1, float * in2, float * out, int N, int totalThreads) {

	int id = (blockIdx.x * blockDim.x) + threadIdx.x;

	while (id < N) {
        float t1 = in1[id];
        float t2 = in2[id];
        float tt = t1 + t2;
		for (int i = 0; i < 10; i++)
			tt *= 3.14 * 2.718 / .57721 - 4.6692 + 1.61803 * 131.7 - 530.1874 / 51.9;
		
		out[id] = tt;
        id += totalThreads;
	}
}

// MAIN FUNCTION CALL TO GET DEVICE-SPECIFIC DATA
void initDeviceVars() {

	getGPU();
    getMaxBlocksPerSM();
    getMaxWarpsPerSM();
}

// SET DEVICE WITH HIGHEST COMPUTE CAPABILITY
void getGPU() {

    int dev_count, deviceToUse, maxCCmajor, maxCCminor;
    dev_count = deviceToUse = maxCCmajor = maxCCminor = 0;
    
    // GET NUMBER OF DEVICES
    hipDeviceProp_t dev_prop;
    hipGetDeviceCount(&dev_count);
    if (dev_count < 1) {
        printf("No CUDA enabled devices on this system!\n");
        exit(1);
    }

    // WHICH DEVICE HAS HIGHEST COMPUTE CAPABILITY
    for (int i = 0; i < dev_count; i++) {
        hipGetDeviceProperties(&dev_prop, i);
        if ((dev_prop.major > maxCCmajor) || ((dev_prop.major == maxCCmajor) && (dev_prop.minor > maxCCminor))) {
            deviceToUse = i;
            maxCCmajor = dev_prop.major;
    		maxCCminor = dev_prop.minor;
        }
    }

    // SET DEVICE/DEVICE-SPECIFIC VARIABLES
    hipGetDeviceProperties(&dev_prop, deviceToUse);
    hipSetDevice(deviceToUse);
    deviceName = &dev_prop.name[0];
    compCapMajor = maxCCmajor;
    compCapMinor = maxCCminor;
    maxThreadsPerGrid = dev_prop.maxGridSize[0];
    numSMs = dev_prop.multiProcessorCount;
    maxThreadsPerBlock = dev_prop.maxThreadsPerBlock;
}

// GET MAX NUMBER OF BLOCKS ASSIGNABLE TO AN SM
void getMaxBlocksPerSM() {

	if (compCapMajor == 2) maxBlocksPerSM = 8;
	else if (compCapMajor == 3) maxBlocksPerSM = 16;
	else if ((compCapMajor == 5) || (compCapMajor == 6)) maxBlocksPerSM = 32;
	else {
		printf("\n No max blocks settings for Compute Capability %d.%d\n", 
			compCapMajor, compCapMinor);
		exit(0);
	}
}

// GET MAX NUMBER OF WARPS AND THREADS THAT CAN RUN ON AN SM
void getMaxWarpsPerSM() {

	if (compCapMajor == 2) maxWarpsPerSM = 48;
	else if ((compCapMajor == 3) || (compCapMajor == 5)) maxWarpsPerSM = 64;
	else if (compCapMajor == 6) {
		if (compCapMinor == 2) maxWarpsPerSM = 128;
		else maxWarpsPerSM = 64;
	}
	else {
		printf("\n No max warp settings for Compute Capability %d.%d\n", 
			compCapMajor, compCapMinor);
		exit(0);
	}
	// ASSIGN MAX THREADS PER SM
	maxThreadsPerSM = (maxWarpsPerSM * 32);
}

void howToUse() {

    fprintf( stderr, "\nUsage: './occupancy [occupancyMethod] [functionToUse] [targetOccupancy] [problemSize]'");
    fprintf( stderr, "\n\tOccupancy Method:\n\t0: %% of max blocks that can run simultaneously\n\t1: %% of max threads per block\n\t2: inversely scale number of blocks with threads per block");
    fprintf( stderr, "\n\tFunction to Use:\n\t0: doubleInt\n\t1: vectorMaths1\n\t2:vectorMaths2");
    fprintf( stderr, "\n\n\tIE: './occupancy 0 0 75 100000' runs the kernel with doubleInt() and 75%% of max blocks simultaneously assignable to all SMs and a problem size of 100,000");

    exit( 1 );
}