#include "hip/hip_runtime.h"
/* 
 *  
 *  New York University
 *  GPUs Fall 2017
 *  Steven Adam & Michael Corso
 *  Quantifying the Relationship Between Occupancy and Performance
 * 
 *
 *  Code Explanation:
 *
 *      (1) The first function call is to initDeviceVars() which checks the system for CUDA devices and
 *      chooses the device with the highest Compute Capability (CC). Device-specific variables are then 
 *      populated based on the CC and variables stored in the hipDeviceProp_t struct. This information
 *      allows for grid and block dimensions to be constructed according to the specific device this code
 *      is running on.
 *
 *      (2) The code then scales the kernel's parameters based on the device's specifications using the 
 *		user-specified values provided in the program's arguments:
 *
 *          (a) occupancyMethod: 
 *
 *              (0) Blocks per SM: Determines the maximum number of blocks assignable (IE [number of SMs] *
 *              [max blocks assignable to each SM]) and scales it based on the specified targetOccupancy.
 *              The number of threads per block is maxed.
 *
 *              (1) Threads per Block: Determines the maximum number of threads assignable to a block (IE
 *              1024 is common) and scales this based on the specified targetOccupancy. The number of blocks
 *              is equal to ([number of SMs] * [max blocks assignable to each SM]).
 *
 *      	(b) The work being performed by the threads is user-specified in the program's arguments:
 *      
 *          	(0) doubleInt(): 	No memory accesses, simply multiples its thread id by 2
 *
 *          	(1) memoryBound():  Memory-bound vector addition, 3 memory accesses, 1 floating-point addition
 *								    CGMA = 1/3
 *              
 *          	(2) computeBound(): Compute-bound vector math, 3 memory accesses, 90 floating-point operations
 *									CGMA = 90/3 = 30
 *
 *          (c) targetOccupancy: An integer value of 1 - 100 which specifies the percentage of the maximum 
 *              occupancy for this test.
 *              
 *          (d) problemSize: An integer value which specifies the amount of work to be performed by the kernel
 *              
 *
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
#include <algorithm>
#include <vector>

// DEBUG/TEST
#define TESTING false
#define PRINTTIME false

//
void howToUse();

// OCCUPANCY FUNCTIONS
double test_BlocksPerGrid();
double test_ThreadsPerBlock();

// GPU SPEC FUNCTIONS
void initDeviceVars();
void getGPU();
void getMaxBlocksPerSM();
void getMaxWarpsPerSM();

// GPU FUNCTIONS
__global__
void doubleInt (int, int);
__global__
void memoryBound (float *, float *, float *, int, int);
__global__
void computeBound (float *, float *, float *, int, int);

// TEST VARIABLES
int problemSize, occupancyMethod, functionToUse;
double targetOccupancy;

// DEVICE VARIABLES
char * deviceName;
int maxThreadsPerBlock, 
	maxThreadsPerSM, maxBlocksPerSM, maxWarpsPerSM, numSMs, 
	maxThreadsPerGrid, 
	compCapMajor, compCapMinor;

int main(int argc, char * argv[]) {

	// HOW TO USE
    if(argc != 5) howToUse();

	// UPDATE DEVICE VARIABLES
	initDeviceVars();

    // OUTPUT DEVICE-SPECIFIC VALUES
    if (TESTING) {
        printf("\nGPU Info:\n\t%-15s %s\n\t%-15s %d.%d\n\t%-15s %d\n\t%-15s %d\n\t%-15s %d\n\t%-15s %d\n", 
            "Device ID", deviceName,
            "Compute C.", compCapMajor, compCapMinor, 
            "Grid Size", maxThreadsPerGrid,
            "Block Size", maxThreadsPerBlock,
            "# SMs", numSMs,
            "# Warps", maxWarpsPerSM
            );
    }

    // GET USER-SPECIFIED VARIABLES 
    occupancyMethod = (int) atoi(argv[1]);
    functionToUse = (int) atoi(argv[2]);
    if (functionToUse > 2)
        howToUse();
    targetOccupancy = ((double) (atoi(argv[3]) / 100.0));
    if (targetOccupancy > 1.0) targetOccupancy = 1.0;
    if (targetOccupancy == 0.0) targetOccupancy = 0.01;
    problemSize = (int) atoi(argv[4]);

    // MAX BLOCKS THAT CAN RUN SIMULTANEOUSLY
    if (occupancyMethod == 0) {
        test_BlocksPerGrid();
    }
    // MAX THREADS PER BLOCK
    else if (occupancyMethod == 1) {
        test_ThreadsPerBlock();
    }
    else {
        printf("\nNot an acceptable occupancyMethod!\n");
        howToUse();
    }

    return 0;
}

// BLOCKS PER SM / TOTAL BLOCKS IN THE KERNEL (USES MAX NUMBER OF THREADS PER BLOCK)
double test_BlocksPerGrid() {

	// NUMBER OF BLOCKS
    int totalBlocks = ((numSMs * maxBlocksPerSM) * targetOccupancy);
    if (totalBlocks < 1) totalBlocks = 1;
    
    // ATTEMPT TO DISTRIBUTE THREADS EVENLY 
    int threadsPerBlock = (maxThreadsPerSM * (numSMs / totalBlocks));
    while (threadsPerBlock % 32 != 0) threadsPerBlock -= 1;
    if (threadsPerBlock > maxThreadsPerBlock) threadsPerBlock = maxThreadsPerBlock;
    if (threadsPerBlock < 1) threadsPerBlock = 32;

    // TOTAL NUMBER OF THREADS IN THE GRID
    int totalThreads = totalBlocks * threadsPerBlock;

    dim3 dimGrid(totalBlocks, 1, 1);                       
    dim3 dimBlock(threadsPerBlock, 1, 1);

    if (TESTING) printf("\ntest_MaxBlocksPerSM running with:\n\ttotalBlocks\t%d\t%d%%\n\tblockSize\t%d\t%.01f%%\n", 
        totalBlocks, ((int) (targetOccupancy * 100)), threadsPerBlock, (((float) threadsPerBlock / (float) maxThreadsPerBlock) * 100));

    // ARRAYS FOR PERFORMING VECTOR MATH
    float * in1 = (float *) calloc((problemSize), sizeof(float));
    float * in2 = (float *) calloc((problemSize), sizeof(float));
    float * out = (float *) calloc((problemSize), sizeof(float));
    float * in1D; float * in2D; float * outD;

    for (int i = 0; i < problemSize; i++) {
        in1[i] = (i * 0.99);
        in2[i] = ((problemSize - i - 1) * 0.99);
        out[i] = -1;
    }

    hipMalloc((void **) &in1D, problemSize);  
    hipMemcpy(in1D, in1, problemSize, hipMemcpyHostToDevice);   
    hipMalloc((void **) &in2D, problemSize);  
    hipMemcpy(in2D, in2, problemSize, hipMemcpyHostToDevice);  
    hipMalloc((void **) &outD, problemSize);  
    hipMemcpy(outD, out, problemSize, hipMemcpyHostToDevice);  

    // INITIALIZE TIMER BEFORE CALLING KERNEL
    // clock_t start = clock();
    if (functionToUse == 0) {
        doubleInt<<<dimGrid, dimBlock>>>(problemSize, totalThreads);
    }
    else if (functionToUse == 1) {
        memoryBound<<<dimGrid, dimBlock>>>(in1D, in2D, outD, problemSize, totalThreads);
    }
    else if (functionToUse == 2) {
        computeBound<<<dimGrid, dimBlock>>>(in1D, in2D, outD, problemSize, totalThreads);
    }

    // SYNC DEVICE AND GET TIME TAKEN
    hipDeviceSynchronize();
    // clock_t end = clock();
    // double time_taken = ((double)(end - start)) / CLOCKS_PER_SEC;

    // CLEANUP
    free(in1); free(in2); free(out);
    hipFree(in1D); hipFree(in2D); hipFree(outD);

    return time_taken;
}

// THREADS PER BLOCK (USES MAX NUMBER OF BLOCKS)
double test_ThreadsPerBlock() {

	// NUMBER OF THREADS PER BLOCK
    int threadsPerBlock = (maxThreadsPerBlock * targetOccupancy);
    while (threadsPerBlock % 32 != 0) threadsPerBlock -= 1;
    if (threadsPerBlock < 1) threadsPerBlock = 1;

    // NUMBER OF BLOCKS
    int totalBlocks = (maxThreadsPerSM / threadsPerBlock) * numSMs;
    if (totalBlocks < 1) totalBlocks = 1;

    // TOTAL NUMBER OF THREADS IN THE GRID
    int totalThreads = totalBlocks * threadsPerBlock;

    dim3 dimGrid(totalBlocks, 1, 1);                       
    dim3 dimBlock(threadsPerBlock, 1, 1);

    if (TESTING) printf("\ntest_ThreadsPerBlock running with:\n\ttotalBlocks\t%d\t100%%\n\tblockSize\t%d\t%d%%\n", 
        totalBlocks, threadsPerBlock, ((int) (targetOccupancy * 100)));

    // ARRAYS FOR PERFORMING VECTOR MATH
    float * in1 = (float *) calloc((problemSize), sizeof(float));
    float * in2 = (float *) calloc((problemSize), sizeof(float));
    float * out = (float *) calloc((problemSize), sizeof(float));
    float * in1D; float * in2D; float * outD;

    for (int i = 0; i < problemSize; i++) {
        in1[i] = (i * 0.99);
        in2[i] = ((problemSize - i - 1) * 0.99);
        out[i] = -1;
    }

    hipMalloc((void **) &in1D, problemSize);  
    hipMemcpy(in1D, in1, problemSize, hipMemcpyHostToDevice);   
    hipMalloc((void **) &in2D, problemSize);  
    hipMemcpy(in2D, in2, problemSize, hipMemcpyHostToDevice);  
    hipMalloc((void **) &outD, problemSize);  
    hipMemcpy(outD, out, problemSize, hipMemcpyHostToDevice);  

    // INITIALIZE TIMER BEFORE CALLING KERNEL
    clock_t start = clock();
    if (functionToUse == 0) {
        doubleInt<<<dimGrid, dimBlock>>>(problemSize, totalThreads);
    }
    else if (functionToUse == 1) {
        memoryBound<<<dimGrid, dimBlock>>>(in1D, in2D, outD, problemSize, totalThreads);
    }
    else if (functionToUse == 2) {
        computeBound<<<dimGrid, dimBlock>>>(in1D, in2D, outD, problemSize, totalThreads);
    }

    // SYNC DEVICE AND GET TIME TAKEN
    hipDeviceSynchronize();
    clock_t end = clock();
    double time_taken = ((double)(end - start)) / CLOCKS_PER_SEC;

    // CLEANUP
    free(in1); free(in2); free(out);
    hipFree(in1D); hipFree(in2D); hipFree(outD);

    return time_taken;
}

// SIMPLE FP MULTIPLICATION, NO MEMORY ACCESS
__global__
void doubleInt (int N, int totalThreads) {

	int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    int val = id;

	while (id < N) {
        val = id;
        val *= 2;
        id += totalThreads;
	}
}

// MEMORY BOUND VECTOR
__global__
void memoryBound (float * in1, float * in2, float * out, int N, int totalThreads) {

	int id = (blockIdx.x * blockDim.x) + threadIdx.x;

	while (id < N) {
		float t1 = in1[id];
		float t2 = in2[id];
		
		out[id] = t1 + t2;

        id += totalThreads;
	}
}

__global__
void computeBound (float * in1, float * in2, float * out, int N, int totalThreads) {

	int id = (blockIdx.x * blockDim.x) + threadIdx.x;

	while (id < N) {
        float t1 = in1[id];
        float t2 = in2[id];
        float tt = t1 + t2;
		for (int i = 0; i < 10; i++)
			tt *= 3.14 * 2.718 / .57721 - 4.6692 + 1.61803 * 131.7 - 530.1874 / 51.9;
		
		out[id] = tt;
        id += totalThreads;
	}
}

// MAIN FUNCTION CALL TO GET DEVICE-SPECIFIC DATA
void initDeviceVars() {

	getGPU();
    getMaxBlocksPerSM();
    getMaxWarpsPerSM();
}

// SET DEVICE WITH HIGHEST COMPUTE CAPABILITY
void getGPU() {

    int dev_count, deviceToUse, maxCCmajor, maxCCminor;
    dev_count = deviceToUse = maxCCmajor = maxCCminor = 0;
    
    // GET NUMBER OF DEVICES
    hipDeviceProp_t dev_prop;
    hipGetDeviceCount(&dev_count);
    if (dev_count < 1) {
        printf("No CUDA enabled devices on this system!\n");
        exit(1);
    }

    // WHICH DEVICE HAS HIGHEST COMPUTE CAPABILITY
    for (int i = 0; i < dev_count; i++) {
        hipGetDeviceProperties(&dev_prop, i);
        if ((dev_prop.major > maxCCmajor) || ((dev_prop.major == maxCCmajor) && (dev_prop.minor > maxCCminor))) {
            deviceToUse = i;
            maxCCmajor = dev_prop.major;
    		maxCCminor = dev_prop.minor;
        }
    }

    // SET DEVICE/DEVICE-SPECIFIC VARIABLES
    hipGetDeviceProperties(&dev_prop, deviceToUse);
    hipSetDevice(deviceToUse);
    deviceName = &dev_prop.name[0];
    compCapMajor = maxCCmajor;
    compCapMinor = maxCCminor;
    maxThreadsPerGrid = dev_prop.maxGridSize[0];
    numSMs = dev_prop.multiProcessorCount;
    maxThreadsPerBlock = dev_prop.maxThreadsPerBlock;
}

// GET MAX NUMBER OF BLOCKS ASSIGNABLE TO AN SM
void getMaxBlocksPerSM() {

	if (compCapMajor == 2) maxBlocksPerSM = 8;
	else if (compCapMajor == 3) maxBlocksPerSM = 16;
	else if ((compCapMajor == 5) || (compCapMajor == 6)) maxBlocksPerSM = 32;
	else {
		printf("\n No max blocks settings for Compute Capability %d.%d\n", 
			compCapMajor, compCapMinor);
		exit(0);
	}
}

// GET MAX NUMBER OF WARPS AND THREADS THAT CAN RUN ON AN SM
void getMaxWarpsPerSM() {

	if (compCapMajor == 2) maxWarpsPerSM = 48;
	else if ((compCapMajor == 3) || (compCapMajor == 5)) maxWarpsPerSM = 64;
	else if (compCapMajor == 6) {
		if (compCapMinor == 2) maxWarpsPerSM = 128;
		else maxWarpsPerSM = 64;
	}
	else {
		printf("\n No max warp settings for Compute Capability %d.%d\n", 
			compCapMajor, compCapMinor);
		exit(0);
	}
	// ASSIGN MAX THREADS PER SM
	maxThreadsPerSM = (maxWarpsPerSM * 32);
}

void howToUse() {

    fprintf( stderr, "\nUsage: './occupancy [occupancyMethod] [functionToUse] [targetOccupancy] [problemSize]'");
    fprintf( stderr, "\n\tOccupancy Method:\n\t\t0: %% of max blocks that can run simultaneously\n\t\t1: %% of max threads per block");
    fprintf( stderr, "\n\tFunction to Use:\n\t\t0: doubleInt\n\t\t1: memoryBound\n\t\t2: computeBound");
    fprintf( stderr, "\n\n\tIE: './occupancy 0 0 75 100000' runs the kernel with doubleInt() and 75%% of max blocks simultaneously assignable to all SMs and a problem size of 100,000");

    exit( 1 );
}